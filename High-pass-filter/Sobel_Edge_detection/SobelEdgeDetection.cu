#include "hip/hip_runtime.h"
#include <iostream>
#include <FreeImage.h>
#include <time.h>

#define THR 50
#define MAX 255
#define MIN 0

#define MASK_DIM 3

#define MASK_OFFSET (MASK_DIM / 2)

__constant__ int  mask[MASK_DIM * MASK_DIM];

__global__ void RgbToGrey(const BYTE* in, BYTE* out, int width, int height)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    float   temp_r = 0.0f;
    float   temp_g = 0.0f;
    float   temp_b = 0.0f;

    // Check if the thread is within the image dimensions
    if (row < height && col < width)
    {

	int start_r  = row - MASK_OFFSET;
	int start_c  = col - MASK_OFFSET;
        int pixelIndex = row * width + col;
	


  	for (int i = -1; i <= 1; i++) {
    	// Go over each column
    	for (int j = -1; j <= 1; j++) {
          temp_r		+= in[((i + row) * width + (j + col)) * 3] * mask[(i + 1)  * MASK_DIM + (j + 1) ];
          temp_g		+= in[((i + row) * width + (j + col)) * 3 + 1] * mask[(i + 1) * MASK_DIM + (j + 1)];
          temp_b 		+= in[((i + row) * width + (j + col)) * 3 + 2] * mask[((i + 1)  * MASK_DIM + (j + 1))];

		}
    }
  


  	// Write back the result
  	out[(row * width + col) * 3] = temp_r;
  	out[(row * width + col) * 3 + 1] = temp_g;
  	out[(row * width + col) * 3 + 2] = temp_b;

    } 
}

void checkError(hipError_t error)
{
    if (error != hipSuccess)
    {
        std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl;
        exit(EXIT_FAILURE);
    }
}

int main()
{
    const char* inputFileName = "input.bmp";
    const char* outputFileName = "Sobel.bmp";

     struct timespec currentTime1,currentTime2;

    // Load input BMP image using FreeImage
    FIBITMAP* inputImage = FreeImage_Load(FIF_BMP, inputFileName, BMP_DEFAULT);
    if (!inputImage)
    {
        std::cerr << "Error loading input image" << std::endl;
        return EXIT_FAILURE;
    }



    int horizontal[9] = {-1,-2,-1,0,0,0,1,2,1};
    int vertical[9] = {-1,0,1,-2,0,2,-1,0,1}; 

     int ch;


     //int *h_mat = (int*)malloc(9 * sizeof(int));

     int h_mat[9];
     printf("1. Horizontal\n2. Vertical Edge detection\n Enter your  choice\n");
     scanf("%d",&ch);

     switch(ch){
	     case 1: 
		     memcpy((void*)&h_mat,(void*)&horizontal, (9 * sizeof(int)));
		     break;
	     case 2:
		     memcpy((void*)&h_mat,(void*)&vertical, (9 * sizeof(int)));
		     break;

     }






    // Get image dimensions
    int width = FreeImage_GetWidth(inputImage);
    int height = FreeImage_GetHeight(inputImage);

    // Convert image to 24-bit RGB format
    FIBITMAP* inputImageRGB = FreeImage_ConvertTo24Bits(inputImage);

    // Allocate memory for GPU buffers
    BYTE* d_input, *d_output;
    checkError(hipMalloc((void**)&d_input, width * height * 3));
    checkError(hipMalloc((void**)&d_output, width * height * 3));

    // Copy input image data to GPU
    checkError(hipMemcpy(d_input, FreeImage_GetBits(inputImageRGB), width * height * 3, hipMemcpyHostToDevice));



    // Size of the mask in bytes
    size_t bytes_m = (MASK_DIM * MASK_DIM * 4);
    
    clock_gettime(CLOCK_REALTIME, &currentTime1);
    long milliseconds1 = currentTime1.tv_nsec / 1000000;
    long seconds1 = currentTime1.tv_sec * 1000;

    dim3 blockDim(16, 16); // Adjust block dimensions as needed
    dim3 gridDim((width + blockDim.x - 1) / blockDim.x, (height + blockDim.y - 1) / blockDim.y);

    hipMemcpyToSymbol(HIP_SYMBOL(mask), h_mat, bytes_m);


    RgbToGrey<<<gridDim, blockDim>>>(d_input, d_output, width, height);

    checkError(hipGetLastError());
    checkError(hipDeviceSynchronize());


    clock_gettime(CLOCK_REALTIME, &currentTime2);
    long milliseconds2 = currentTime2.tv_nsec / 1000000;
    long seconds2 = currentTime2.tv_sec * 1000;

        printf("difference in time is %ld \n",((seconds2+milliseconds2) - (seconds1+milliseconds1)));
    // Copy result from GPU to host
    BYTE* h_output = (BYTE*)malloc(width * height * 3);
    checkError(hipMemcpy(h_output, d_output, width * height * 3, hipMemcpyDeviceToHost));


    // Save output image using FreeImage
    FIBITMAP* outputImage = FreeImage_ConvertFromRawBits(h_output, width, height, 3 * width, 24, 0xFF0000, 0x00FF00, 0x0000FF, false);
    FreeImage_Save(FIF_BMP, outputImage, outputFileName);

    // Cleanup
    hipFree(d_input);
    hipFree(d_output);
    FreeImage_Unload(inputImage);
    FreeImage_Unload(inputImageRGB);
    free(h_output);

    return EXIT_SUCCESS;
}
